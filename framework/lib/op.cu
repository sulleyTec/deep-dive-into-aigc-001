#include "hip/hip_runtime.h"
#include "op.hpp"

namespace geefer {

template<typename DType>
__global__ void vec_add_kernel(const DType *input1, 
                               const DType *input2, 
                               DType *output, 
                               const uint32_t size)
{
    int idx = blockDim.x*blockIdx.x+threadIdx.x;
    if (idx < size)
        output[idx] = input1[idx] + input2[idx];
}

template<typename DType>
void vec_add(const DType *input1, const DType *input2, 
             DType *output, const uint32_t size) {

    int block_size = 256;
    int grid = (size+block_size-1)/block_size;

    vec_add_kernel<<<grid, block_size>>>(input1, input2, output, size);
}

/*  */
template __global__ void vec_add_kernel<float>(const float* input1, 
                                               const float* input2, 
                                               float* b,
                                               const uint32_t size);

template void vec_add<float>(const float* input1, 
                             const float* input2, 
                             float* b,
                             const uint32_t size);

} // namespace geefer



